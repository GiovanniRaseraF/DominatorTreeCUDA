
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>

__global__ void mallocTest(){
    size_t size = 123;
    char* ptr = (char*) malloc(size);

    memset(ptr, 0, size);

    printf("Thread %d pointing to %p", threadIdx.x, ptr);
    //std::cout << "Thread " << threadIdx.x << " got pointer " << ptr << std::endl;

    free(ptr);
}

int main(){
    hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
    mallocTest<<<1, 5>>>();
    hipDeviceSynchronize();
    return 0;
}
