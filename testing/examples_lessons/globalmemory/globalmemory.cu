
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

// Author: Giovanni Rasera

#define N 100
#define NumThPerBlock 256
#define NumBlocks 1

__global__ void vector_sum(int *A, int *B, int *C){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if(thread_id < N) C[thread_id] = A[thread_id] + B[thread_id];
}

#ifdef MAPPED
    #define MTYPE cudaHostAllocMapped
#else
    #define MTYPE hipHostMallocWriteCombined
#endif

int main(){
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // static memory allocation
    hipHostAlloc(&dev_a, N*sizeof(int), MTYPE);
    hipHostAlloc(&dev_b, N*sizeof(int), MTYPE);
    hipHostAlloc(&dev_c, N*sizeof(int), MTYPE);

    // host inits values
    for(int i = 0; i < N; i++){
        dev_a[i] = -i;
        dev_b[i] = i * i;
    }

    // copy memory to gpu
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // run code on gpu
    vector_sum<<<NumBlocks, NumThPerBlock>>>(dev_a, dev_b, dev_c);

    // read result from gpu to cpu
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // results
    std::cout << "[";
    for(int i = 0; i < N-1; i++){
        std::cout << c[i] << ", ";
    }
    std::cout << c[N-1];
    std::cout << "]";
}