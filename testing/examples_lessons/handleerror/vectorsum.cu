
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>

// Author: Giovanni Rasera

#define NumThPerBlock 256
#define NumBlocks 256
#define NumWarp 32
#define N ((NumWarp) * (NumBlocks) * (NumThPerBlock))

static void HandleError(hipError_t err, const char *file, int line){
    if(err != hipSuccess){
        std::cerr << (hipGetErrorString(err)) << " in " << file << " at line " << line << std::endl;
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void vector_sum(long long *A, long long *B, long long *C){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if(thread_id < N) C[thread_id] = A[thread_id] + B[thread_id];
}

int main(){
    long long a[N], b[N], c[N];
    long long *dev_a, *dev_b, *dev_c;

    // static memory allocation
    HANDLE_ERROR(hipMalloc((void**)&dev_a, N*sizeof(long long)));
    hipMalloc((void**)&dev_b, N*sizeof(long long));
    hipMalloc((void**)&dev_c, N*sizeof(long long));

    // host inits values
    for(long long i = 0; i < N; i++){
        a[i] = i+1;
        b[i] = i+1;
    }

    // copy memory to gpu
    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(long long), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(long long), hipMemcpyHostToDevice));

    // run code on gpu
    vector_sum<<<NumBlocks, NumThPerBlock>>>(dev_a, dev_b, dev_c);

    // read result from gpu to cpu
    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(long long), hipMemcpyDeviceToHost));

    // results
    std::cout << "[";
    for(int i = 0; i < N-1; i++){
        if(i % 1000 == 0) std::cout << c[i] << ", " << std::endl;
    }
    std::cout << c[N-1];
    std::cout << " ]" << std::endl;;

}