#include "hip/hip_runtime.h"
#include "book.h"
#include <iostream>
#include <iomanip>

constexpr int NumThPerBlock = 256;
constexpr int NumBlocks = 256;
constexpr int N = NumBlocks * NumThPerBlock;

__global__ void vetadd(float *A, float *B, float *C){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // need to check if i is bigger than the vector length
    if (i < N)
        // this divergence is good because we have 1 block
        C[i] = A[i]+ B[i];
}

int main(){
    std::cout << "CUDA: mat add" << std::endl;
    //on host
    float h_A[N], h_B[N], h_C[N];

    // on device
    // device memory cannot be deferenced
    // so we nell hipMalloc and hipMemcpy 
    // to know result and allocate memory
    float *A, *B, *C;

    // init
    for(int i = 0; i < N; i++){
        h_A[i] = -i;
        h_B[i] = i*i;
        h_C[i] = 0;
    }
    
    // malloc mem on device
    // the memory is STATIC in the sense that
    // the memory is copied before the kernel 
    // execution
    HANDLE_ERROR(
        hipMalloc(
            (void**)&A,
            N*sizeof(float))
    );
    HANDLE_ERROR(
        hipMalloc(
            (void**)&B,
            N*sizeof(float))
    );
    HANDLE_ERROR(
        hipMalloc(
            (void**)&C,
            N*sizeof(float))
    );

    // copy
    HANDLE_ERROR(
        hipMemcpy(
            A,
            h_A,
            N*sizeof(float),
            hipMemcpyHostToDevice
        )
    );
    HANDLE_ERROR(
        hipMemcpy(
            B,
            h_B,
            N*sizeof(float),
            hipMemcpyHostToDevice
        )
    );
    HANDLE_ERROR(
        hipMemcpy(
            C,
            h_C,
            N*sizeof(float),
            hipMemcpyHostToDevice
        )
    );

    // create blocks of threads

    // execute kernel 
    vetadd<<<NumBlocks, NumThPerBlock>>>(A, B, C);

    // retreve result   
    HANDLE_ERROR(
        hipMemcpy(
            h_C,
            C,
            N*sizeof(float),
            hipMemcpyDeviceToHost
        )
    );

    //for(int i = 0; i < N; i++){
        //std::cout << std::setw(4) << std::left << h_A[i] << " + " << std::setw(5) << std::right << h_B[i] << ": " << h_C[i] << std::endl;
    //}

    hipFree(C);
    hipFree(A);
    hipFree(B);

    return 0;
}