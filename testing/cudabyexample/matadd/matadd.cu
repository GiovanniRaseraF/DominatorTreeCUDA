#include "hip/hip_runtime.h"
#include "book.h"
#include <iostream>
#include <iomanip>

constexpr int N = 32*4;

__global__ void matadd(float *A, float *B, float *C){
    int i = threadIdx.x;
    //int j = threadIdx.y;

    C[i] = A[i]+ B[i];
}

int main(){
    std::cout << "CUDA: mat add" << std::endl;
    //on host
    float h_A[N];
    float h_B[N];
    float h_C[N];

    
    // on device
    float *A, *B, *C;

    // init
    for(int i = 0; i < N; i++){
        h_A[i] = 1;
        h_B[i] = 1;
        h_C[i] = 0;
    }
    
    // malloc mem on device
    HANDLE_ERROR(
        hipMalloc(
            (void**)&A,
            N*sizeof(float))
    );
    HANDLE_ERROR(
        hipMalloc(
            (void**)&B,
            N*sizeof(float))
    );
    HANDLE_ERROR(
        hipMalloc(
            (void**)&C,
            N*sizeof(float))
    );

    // copy
    HANDLE_ERROR(
        hipMemcpy(
            A,
            h_A,
            N*sizeof(float),
            hipMemcpyHostToDevice
        )
    );
    HANDLE_ERROR(
        hipMemcpy(
            B,
            h_B,
            N*sizeof(float),
            hipMemcpyHostToDevice
        )
    );
    HANDLE_ERROR(
        hipMemcpy(
            C,
            h_C,
            N*sizeof(float),
            hipMemcpyHostToDevice
        )
    );

    // create blocks of threads
    int numBlocks = 1;
    dim3 threadPerBlock(N);

    // execute kernel 
    matadd<<<numBlocks, threadPerBlock>>>(A, B, C);

    // retreve result   
    HANDLE_ERROR(
        hipMemcpy(
            h_C,
            C,
            N*sizeof(float),
            hipMemcpyDeviceToHost
        )
    );

    for(int i = 0; i < N; i++){
        std::cout << std::setw(3) << h_C[i];
    }

    hipFree(C);
    hipFree(A);
    hipFree(B);

    return 0;
}