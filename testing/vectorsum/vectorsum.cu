
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>

// Author: Giovanni Rasera

#define N 10000
#define NumThPerBlock 256
#define NumBlocks 256

__global__ void vector_sum(long long *A, long long *B, long long *C){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if(thread_id < N) C[thread_id] = A[thread_id] + B[thread_id];
}

int main(){
    long long a[N], b[N], c[N];
    long long *dev_a, *dev_b, *dev_c;

    // static memory allocation
    hipMalloc((void**)&dev_a, N*sizeof(long long));
    hipMalloc((void**)&dev_b, N*sizeof(long long));
    hipMalloc((void**)&dev_c, N*sizeof(long long));

    // host inits values
    for(long long i = 0; i < N; i++){
        a[i] = i+1;
        b[i] = i+1;
    }

    // copy memory to gpu
    hipMemcpy(dev_a, a, N * sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(long long), hipMemcpyHostToDevice);

    // run code on gpu
    vector_sum<<<NumBlocks, NumThPerBlock>>>(dev_a, dev_b, dev_c);

    // read result from gpu to cpu
    hipMemcpy(c, dev_c, N * sizeof(long long), hipMemcpyDeviceToHost);

    // results
    std::cout << "[";
    for(int i = 0; i < N-1; i++){
        if(i % 1000 == 0) std::cout << c[i] << ", " << std::endl;
    }
    std::cout << c[N-1];
    std::cout << " ]" << std::endl;;

}