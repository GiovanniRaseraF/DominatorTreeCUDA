#include <iostream>
#include <iomanip>

int main(){
    std::cout << "N: " << N << " float allocation"<< std::endl;
    //on host
    float h_A[N];
    float *A, *B, *C;

    for(int i = 2; i < 1024 * 512 - N; i*=2){
        hipMalloc(
            (void**)&A,
            N*sizeof(float));

        // copy
        hipMemcpy(
            A,
            h_A,
            N*sizeof(float),
            hipMemcpyHostToDevice
        );

        // retreve result   
        hipMemcpy(
            h_A,
            A,
            N*sizeof(float),
            hipMemcpyDeviceToHost
        );

        hipFree(A);
    }

    return 0;
}