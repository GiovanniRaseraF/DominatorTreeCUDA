#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>
#include <iomanip>
#include <algorithm>
#include <numeric>
#include "gpu_reduce.cuh"

float reduce(float *h_Results, float *A, float *Results);

int main(){
    std::cout << "N: " << N << " -" << reduce_type << " reduce"<< std::endl;

    //on host
    float h_A[N], h_Results[N];
    float *A, *Results;

    for(int i = 0; i < N; i++){
        h_A[i] = 1;
        h_Results[i] = 0;
    }

    hipMalloc(
        (void**)&A,
        N*sizeof(float));
    hipMalloc(
        (void**)&Results,
        N*sizeof(float));

    // copy
    hipMemcpy(
        A,
        h_A,
        N*sizeof(float),
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        Results,
        h_Results,
        N*sizeof(float),
        hipMemcpyHostToDevice
    );

    auto sum = reduce(h_Results, A, Results);

    std::cout << "result= " << sum << ", correct?" << std::boolalpha << " " << (sum == N) << std::endl;

    hipFree(A);
    hipFree(Results);

    return 0;
}