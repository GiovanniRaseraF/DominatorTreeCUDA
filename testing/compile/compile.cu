
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <thread>

int main(){
    std::cout << "cuda compile" << std::endl;

    int *a;
    hipMalloc(&a, 10);
    hipFree(a);

    std::cout << "result: " << "passed" << std::endl;

    return 0;
}