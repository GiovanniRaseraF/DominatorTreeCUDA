#include <iostream>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <memory>
#include <thread>

int main(){
    std::cout << "cuda compile" << std::endl;

    int *a;
    hipMalloc(&a, 10);
    hipFree(a);

    #pragma omp parallel
    {
        #pragma omp critical
        {
            std::cout << "thread num: " << omp_get_thread_num() << std::endl;
        }
    }

    std::cout << "result: " << "passed" << std::endl;

    return 0;
}